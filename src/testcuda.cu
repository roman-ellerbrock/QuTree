#include "hip/hip_runtime.h"
#include "cblas.h"
#include "lapacke.h"
#include <hip/hip_runtime.h>
#include <>
#include <hipblas.h>
#include <hiprand.h>
#include <math.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <iostream>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void simple_dgemm(float* C, float* A, float* B, int m) {
    float alpha = 1.0f;
    float beta = 0.0f;
    cblas_sgemm(CblasColMajor, CblasNoTrans, CblasNoTrans, m, m, m, alpha, A, m, B, m, beta, C, m);
}

/*void simple_cublasDgemm(float* C, float* A, float* B, int m) {
        float alpha = 1.;
    float beta = 0.;
    hipblasDgemm(cublasGlobalHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, m, m,
        (float *)gpu_alpha1, (float *)&gpu_A[0], 1024, (float *)&gpu_B
         [0], 1024, (float *)gpu_beta1, (float *)&gpu_C[0], 1024);
}*/

float residual(float* A, float* B, int N) {
    float r = 0.;
    for (int i = 0; i < N; ++i) {
        r += pow(abs(A[i] - B[i]), 2);
    }
    return sqrt(r);
}

int cu_main(int m, int runs) {
    int N = m*m;
    int bytes = N * sizeof(float);

    float *dev_A, *dev_B, *dev_C;
    float* host_A = (float*) malloc(bytes);
    float* host_B = (float*) malloc(bytes);
    float* host_C = (float*) malloc(bytes);
    float* host_C2 = (float*) malloc(bytes);
    hipMalloc(&dev_A, bytes);
    hipMalloc(&dev_B, bytes);
    hipMalloc(&dev_C, bytes);

    // random number generator
    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

    // Set the seed
    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

    // Fill matrices with random numbers on device
    hiprandGenerateUniform(prng, dev_A, N);
    hiprandGenerateUniform(prng, dev_B, N);

    // cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float alpha = 1.0f;
    float beta = 0.0f;
    
    // prepare CUDA event timer
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // call dgemm on Device
    hipEventRecord(start, 0);
    for (size_t i = 0; i < runs; ++i) {
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, m, m, &alpha, dev_A, m, dev_B, m, &beta, dev_C, m);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    gpuErrchk(hipPeekAtLastError());

    // report time
    hipEventElapsedTime(&time, start, stop);
    double d_t = time/= (double) runs;
//    printf ("Time on device: %f ms\n", d_t);

    // copy data back and compare
    hipMemcpy(host_A, dev_A, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(host_B, dev_B, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(host_C, dev_C, bytes, hipMemcpyDeviceToHost);
    
    // prepare CPU timer
    clock_t t;
    t = clock();

    // cal dgemm on Host
    for (size_t i = 0; i < runs; ++i) {
        simple_dgemm(host_C2, host_A, host_B, m);
    }
    t = clock() - t;
    double h_t = ((float)t*1000)/CLOCKS_PER_SEC/(double)runs;
//    printf ("Time on host: %f ms.\n", h_t);

    // verify solution
//    std::cout << "Residual: " << residual(host_C, host_C2, N) << std::endl;

    std::cout << m << " " << h_t << " " << d_t << std::endl;
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);
    return 0;
}

 