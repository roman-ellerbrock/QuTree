#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <stdafx.h>

template <typename TL, typename TR>
__global__ void cudaCast_kernel(TL* dest, const TR* src, size_t n) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        dest[idx] = (float) src[idx];
    }
}

template <typename TL, typename TR>
void cudaCast(TL* dest, const TR* src, size_t n) {
    constexpr size_t nthreads = 1024;
    size_t nblocks = (n + nthreads - 1) / nthreads;
    cudaCast_kernel<<<nblocks, nthreads>>>(dest, src, n);
}

template void cudaCast(float* dest, const double* src, size_t n);

/*__global__ void cuda_diagmm(double* C, const double* da, const double* b, size_t nrow, size_t ncol) {
    row=blockIdx.x*blockDim.x+threadIdx.x;
    col=blockIdx.y*blockDim.y+threadIdx.y;
    if (row < nrow && col < ncol) {

    }
}*/