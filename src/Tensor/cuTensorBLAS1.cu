#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <stdafx.h>

using f = float;
using d = double;
using cf = complex<f>;
using cd = complex<d>;

template <typename TL, typename TR>
__global__ void cudaCast_kernel(TL* dest, const TR* src, size_t n) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        dest[idx] = (TL) src[idx];
    }
}

template <typename TL, typename TR>
void cudaCast(TL* dest, const TR* src, size_t n) {
    constexpr size_t nthreads = 1024;
    size_t nblocks = (n + nthreads - 1) / nthreads;
    cudaCast_kernel<<<nblocks, nthreads>>>(dest, src, n);
}

template void cudaCast(float* dest, const double* src, size_t n);

template <typename T>
__global__ void cudaHadamardProduct_kernel(T* C, const T* A, const T* B, size_t n) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        C[idx] += A[idx] * B[idx];
    }
}

template <typename T>
void cudaHadamardProduct(T* C, const T* A, const T* B, size_t n) {
    constexpr size_t nthreads = 1024;
    size_t nblocks = (n + nthreads - 1) / nthreads;
    cudaHadamardProduct_kernel<<<nblocks, nthreads>>>(C, A, B, n);
}

template void cudaHadamardProduct(d* C, const d* A, const d* B, size_t n);

template <typename T>
__global__ void cudaDiagMatrixMatrix_kernel(T* C, const T* dA, const T* B, T factor, size_t nrow, size_t ncol) {
//    size_t row = blockIdx.x * blockDim.x + threadIdx.x;
//    size_t col = blockIdx.y * blockDim.y + threadIdx.y;
    size_t id_global = blockIdx.x * blockDim.x + threadIdx.x;
    size_t row = id_global % nrow;
    size_t col = id_global / nrow;
    if (row < nrow && col < ncol) {
        size_t idx = row + nrow * col;
        C[idx] += factor * dA[row] * B[idx];
    }
}

template <typename T>
void cudaDiagMatrixMatrix(T* C, const T* dA, const T* B, T factor, size_t nrow, size_t ncol) {
    constexpr size_t nthreads = 128;
//    size_t nblock_row = (nrow + nthreads - 1) / nthreads;
//    size_t nblock_col = (ncol + nthreads - 1) / nthreads;
//    dim3 nblocks3(nblock_row, nblock_col);
//    dim3 nthreads3(nthreads, nthreads);
//    cout << nrow << " " << ncol << endl;
//    cout << nblocks3.x << " " << nblocks3.y << " " << nblocks3.z << endl;
//    cout << nthreads3.x << " " << nthreads3.y << " " << nthreads3.z << endl;
//    cudaDiagMatrixMatrix_kernel<<<nblocks3, nthreads3>>>(C, dA, B, factor, nrow, ncol);
    size_t nblocksX = (nrow * ncol + nthreads - 1) / nthreads;
    dim3 nblocks3(nblocksX, 1);
    dim3 nthreads3(nthreads, 1);
    cudaDiagMatrixMatrix_kernel<<<nblocks3, nthreads3>>>(C, dA, B, factor, nrow, ncol);
}

template void cudaDiagMatrixMatrix(d* C, const d* dA, const d* B, d factor, size_t nrow, size_t ncol);

/*__global__ void cuda_diagmm(double* C, const double* da, const double* b, size_t nrow, size_t ncol) {
    row=blockIdx.x*blockDim.x+threadIdx.x;
    col=blockIdx.y*blockDim.y+threadIdx.y;
    if (row < nrow && col < ncol) {

    }
}*/

